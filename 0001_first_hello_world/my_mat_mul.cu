#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <algorithm>
#include <cassert>
#include <iomanip>

using namespace std;

// 用宏变长参数来实现
#define CUDA_CALL(...) {hipError_t _cuda_tep_set_not_repeat_a=(__VA_ARGS__);if (_cuda_tep_set_not_repeat_a!=hipSuccess){printf("\nCUDA ERROR: %s (err_num=%d)\n", hipGetErrorString(_cuda_tep_set_not_repeat_a), _cuda_tep_set_not_repeat_a); hipDeviceReset(); assert(0);} }
#define CUDA_LAST_ERROR() CUDA_CALL(hipGetLastError())

#define SHOW_MAT(a, m,n) \
{\
  cout<<"\nShowMat: "<<#a<<endl;\
  for (int i=0;i<m;++i){\
    for (int j=0;j<n;++j){\
      cout<<setw(4)<<a[i*n+j];\
    }\
    cout<<endl;\
  }\
}

__global__ void matmult_v1(float *a, float *b, float *c, int m, int n, int k){//a-> m*k  b->k*n

  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  int idy=blockIdx.y*blockDim.y+threadIdx.y;
  int index=idy*n+idx;
  if (idx>=n || idy>=m) return;
  
  c[index]=0;
  for (int i=0;i<k;++i){
    c[index]+=a[idy*k+i]*b[idx+i*n];
  }
}

int main(){
  //获取设备属性
	hipDeviceProp_t prop;
	int deviceID;
	hipGetDevice(&deviceID);
	hipGetDeviceProperties(&prop, deviceID);
 
	//检查设备是否支持重叠功能
	if (!prop.deviceOverlap)
	{
		printf("No device will handle overlaps. so no speed up from stream.\n");
		return 0;
	}

  //启动计时器
	hipEvent_t start, stop;
	float elapsedTime;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);


  int m=28;
  int n=34;
  int k=24;

  float *A=new float[m*k]{1,2,3,4,5,6,7,8,9,10,11};
  float *B=new float[k*n]{1,0,1,0,1,0,0,1,1,0,1,1,0};
  float *C=new float[m*n];

  float *ga, *gb, *gc;
  // GPU端分配内存
  hipMalloc((void**)&ga, m*k*sizeof(float));
  hipMalloc((void**)&gb, k*n*sizeof(float));
  hipMalloc((void**)&gc, m*n*sizeof(float));

  // CPU的数据拷贝到GPU端
  hipMemcpy(ga, A, m*k*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(gb, B, k*n*sizeof(float), hipMemcpyHostToDevice);
  //hipMemcpy(gc, C, size, hipMemcpyHostToDevice);

  // 定义kernel执行配置，（1024*1024/512）个block，每个block里面有512个线程
  dim3 dimBlock(2,3);
  dim3 dimGrid(3,2);

  // 执行kernel
  matmult_v1<<<dimGrid, dimBlock>>>(ga, gb, gc, m,n,k);
  CUDA_LAST_ERROR();

  //hipMemcpy ( void* dst, const void* src, size_t count, hipMemcpyKind kind )
  hipMemcpy(C, gc, m*n*sizeof(float), hipMemcpyDeviceToHost);

  hipFree(ga);
  hipFree(gb);
  hipFree(gc);

  SHOW_MAT(A,m,k);
  SHOW_MAT(B,k,n);
  SHOW_MAT(C,m,n);

  delete []A;
  delete []B;
  delete []C;

  return 0;
}
