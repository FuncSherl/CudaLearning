#include "hip/hip_runtime.h"
#include <algorithm>
#include <iomanip>
#include <iostream>
#include <vector>
#include "commontools.h"

using namespace std;

#define SHOW_MAT(a, m, n)                                           \
  {                                                                 \
    cout << "\nShowMat: " << #a << " -> " << m << "*" << n << endl; \
    for (int i = 0; i < m; ++i) {                                   \
      for (int j = 0; j < n; ++j) {                                 \
        cout << setw(6) << a[i * n + j];                            \
      }                                                             \
      cout << endl;                                                 \
    }                                                               \
  }

#define SET_MAT(a, d, m, n)                            \
  {                                                    \
    cout << "\nsetmat: " << #a << " -> " << d << endl; \
    for (int i = 0; i < m; ++i) {                      \
      for (int j = 0; j < n; ++j) {                    \
        a[i * n + j] = d;                              \
      }                                                \
    }                                                  \
  }

#define SET_MATAUTO(a, m, n)            \
  {                                     \
    cout << "\nsetmat: " << #a << endl; \
    for (int i = 0; i < m; ++i) {       \
      for (int j = 0; j < n; ++j) {     \
        a[i * n + j] = i * n + j;       \
      }                                 \
    }                                   \
  }

#define MDIV(a, b) ((int)a % (int)b == 0 ? (int)a / (int)b : (int)a / (int)b + 1)

template <int BSIZE>
__global__ void mattrans_v1(float *ta, float *a, int m, int n) {
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  int idy = blockIdx.y * blockDim.y + threadIdx.y;
  int ori_ind = idx * n + idy;
  int new_ind = idy * m + idx;
  if (idx >= m || idy >= n) return;

  ta[new_ind] = a[ori_ind];
}

int main() {
  //获取设备属性
  // hipDeviceProp_t prop;
  // int deviceID;
  // hipGetDevice(&deviceID);
  // hipGetDeviceProperties(&prop, deviceID);

  // //对于每个主机线程，每次只有一个 GPU 设备处于活动状态。
  // //如要将特定的 GPU 设置为活动状态，请使用 hipSetDevice 以及所需 GPU 的索引

  // //检查设备是否支持重叠功能
  // //支持设备重叠功能的 GPU 能够在执行一个 CUDA 核函数的同时，还能在主机和设备之间执行复制数据操作
  // if (!prop.deviceOverlap) {
  //   printf("No device will handle overlaps. so no speed up from stream.\n");
  //   return 0;
  // }

  //启动计时器
  hipEvent_t start, stop;
  float elapsedTime;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  int m = 19;
  int n = 23;

  const int BSIZE = 5;

  float *C = new float[m * n];
  SET_MATAUTO(C, m, n);
  SHOW_MAT(C, m, n);

  float *gc, *transgc;
  // GPU端分配内存
  CUDA_CALL(hipMalloc((void **)&gc, m * n * sizeof(float)));
  CUDA_CALL(hipMalloc((void **)&transgc, m * n * sizeof(float)));

  // CPU的数据拷贝到GPU端
  hipMemcpy(gc, C, m * n * sizeof(float), hipMemcpyHostToDevice);
  // hipMemcpy(gc, C, size, hipMemcpyHostToDevice);

  // 定义kernel执行配置，（1024*1024/512）个block，每个block里面有512个线程
  dim3 dimBlock(BSIZE, BSIZE);
  dim3 dimGrid(MDIV(m, BSIZE), MDIV(n, BSIZE));
  cout << "start block num: " << dimGrid.x << "*" << dimGrid.y << endl;
  cout << "each block:" << dimBlock.x << "*" << dimBlock.y << endl;
  // 执行kernel
  CUDA_CALL(hipEventRecord(start));
  int iter = 10 * 200;
  for (int i = 0; i < iter; ++i) {
    mattrans_v1<BSIZE><<<dimGrid, dimBlock>>>(transgc, gc, m, n);
  }
  CUDA_LAST_ERROR();

  CUDA_CALL(hipEventRecord(stop));

  CUDA_CALL(hipEventSynchronize(stop));
  CUDA_CALL(hipEventElapsedTime(&elapsedTime, start, stop));

  // hipMemcpy ( void* dst, const void* src, size_t count, hipMemcpyKind kind )
  hipMemcpy(C, transgc, m * n * sizeof(float), hipMemcpyDeviceToHost);

  CUDA_CALL(hipFree(transgc));
  CUDA_CALL(hipFree(gc));

  SHOW_MAT(C, n, m);
  cout << "Iter:" << iter << " UsedTime: " << elapsedTime << " ms" << endl;
  delete[] C;
  CUDA_CALL(hipEventDestroy(start));
  CUDA_CALL(hipEventDestroy(stop));
  return 0;
}
