#include "hip/hip_runtime.h"
/*
 * @Author: xh
 * @Date: 2020-05-30 15:04:22
 * @LastEditTime: 2020-09-12 14:10:23
 * @LastEditors: Please set LastEditors
 * @Description: In User Settings Edit
 * @FilePath: /workspaces/cuda/cuda_start/plus.cu
 */
#include <algorithm>
#include <iostream>
#include <vector>
#include "commontools.h"

using namespace std;

__global__ void myplus(float a[], float b[], float c[], int n) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  c[i] = a[i] + b[i];
  if (i % 1000 == 0) printf("gpu print:%f\n", c[i]);
}

int main() {
  int n = 1024 * 1024;
  int size = n * sizeof(float);
  float *A = new float[size]{1, 1};
  float *B = new float[size]{2, 2};
  float *C = new float[size]{3, 3};

  float *ga, *gb, *gc;
  // GPU端分配内存
  hipMalloc((void **)&ga, size);
  hipMalloc((void **)&gb, size);
  hipMalloc((void **)&gc, size);

  // CPU的数据拷贝到GPU端
  hipMemcpy(ga, A, size, hipMemcpyHostToDevice);
  hipMemcpy(gb, B, size, hipMemcpyHostToDevice);
  // hipMemcpy(gc, C, size, hipMemcpyHostToDevice);

  // 定义kernel执行配置，（1024*1024/512）个block，每个block里面有512个线程
  dim3 dimBlock(512);
  dim3 dimGrid(n / 512);

  // 执行kernel
  myplus<<<dimGrid, dimBlock>>>(ga, gb, gc, n);

  // hipMemcpy ( void* dst, const void* src, size_t count, hipMemcpyKind kind )
  hipMemcpy(C, gc, size, hipMemcpyDeviceToHost);

  hipFree(ga);
  hipFree(gb);
  hipFree(gc);

  cout << A[0] << "+" << B[0] << "=" << C[0] << endl;
  cout << A[2] << "+" << B[2] << "=" << C[2] << endl;
  delete[] A;
  delete[] B;
  delete[] C;

  return 0;
}
