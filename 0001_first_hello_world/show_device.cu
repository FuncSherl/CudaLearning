
#include <iostream>
#include <stdio.h>
#include "hip/hip_runtime.h"


#define CUDA_CALL(x) {const hipError_t a=(x);if (a!=hipSuccess){printf("\nCUDA ERROR: %s (err_num=%d)\n", hipGetErrorString(a), a); hipDeviceReset(); } }

using namespace std;

int main()
{
    int deviceCount;
    CUDA_CALL( hipGetDeviceCount(&deviceCount));
    std::cout<< "device count: "<<deviceCount<<endl;
    for(int i=0;i<deviceCount;i++)
    {
        hipDeviceProp_t devProp;
        hipGetDeviceProperties(&devProp, i);
        std::cout << "GPU device name: " << i << " : " << devProp.name << std::endl;
        std::cout << "device total Global Mem: " << devProp.totalGlobalMem / 1024 / 1024 << "MB" << std::endl;
        std::cout << "multiProcessor(SM) Count: " << devProp.multiProcessorCount << std::endl;
        std::cout << "shared Mem Per Block: " << devProp.sharedMemPerBlock / 1024.0 << " KB" << std::endl;
        std::cout << "maxThreadsPerBlock: " << devProp.maxThreadsPerBlock << std::endl;
        std::cout << "Registers PerBlock: " << devProp.regsPerBlock << std::endl;
        std::cout << "maxThreads PerMultiProcessor(EM): " << devProp.maxThreadsPerMultiProcessor << std::endl;
        std::cout << "maxWarps PerMultiProcessor: " << devProp.maxThreadsPerMultiProcessor / 32 << std::endl;
        std::cout << "======================================================" << std::endl;             
    }
    return 0;
}
