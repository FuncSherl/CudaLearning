
#include <iostream>
#include <stdio.h>
#include "hip/hip_runtime.h"


#define CUDA_CALL(x) {const hipError_t a=(x);if (a!=hipSuccess){printf("\nCUDA ERROR: %s (err_num=%d)\n", hipGetErrorString(a), a); hipDeviceReset(); } }

using namespace std;

int main()
{
    int deviceCount;
    CUDA_CALL( hipGetDeviceCount(&deviceCount));
    std::cout<< "device count:"<<deviceCount<<endl;
    for(int i=0;i<deviceCount;i++)
    {
        hipDeviceProp_t devProp;
        hipGetDeviceProperties(&devProp, i);
        std::cout << "使用GPU device " << i << ": " << devProp.name << std::endl;
        std::cout << "设备全局内存总量： " << devProp.totalGlobalMem / 1024 / 1024 << "MB" << std::endl;
        std::cout << "SM的数量：" << devProp.multiProcessorCount << std::endl;
        std::cout << "每个线程块的共享内存大小：" << devProp.sharedMemPerBlock / 1024.0 << " KB" << std::endl;
        std::cout << "每个线程块的最大线程数：" << devProp.maxThreadsPerBlock << std::endl;
        std::cout << "设备上一个线程块（Block）种可用的32位寄存器数量： " << devProp.regsPerBlock << std::endl;
        std::cout << "每个EM的最大线程数：" << devProp.maxThreadsPerMultiProcessor << std::endl;
        std::cout << "每个EM的最大线程束数：" << devProp.maxThreadsPerMultiProcessor / 32 << std::endl;
        std::cout << "设备上多处理器的数量： " << devProp.multiProcessorCount << std::endl;
        std::cout << "======================================================" << std::endl;     
        
    }
    return 0;
}
