#include <stdio.h>
#include <cassert>
#include "hip/hip_runtime.h"


#define CUDA_CALL(x) {const hipError_t a=(x);if (a!=hipSuccess){printf("\nCUDA ERROR: %s (err_num=%d)\n", hipGetErrorString(a), a); hipDeviceReset(); assert(0);} }

int main(){
    CUDA_CALL(hipSetDevice(0));
    //init
    const int length=1024;
    int a[length], b[length];
    for (int i=0;i<length;++i){
        a[i]=i;
        b[i]=i*i;
    }
    int *datas=NULL;
    hipMalloc((void **)&datas, length * sizeof(int));



    return 0;
}

