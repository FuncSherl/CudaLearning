#include <stdio.h>
#include <cassert>
#include "hip/hip_runtime.h"


// 用宏变长参数来实现
#define CUDA_CALL(...) {hipError_t _cuda_tep_set_not_repeat_a=(__VA_ARGS__);if (_cuda_tep_set_not_repeat_a!=hipSuccess){printf("\nCUDA ERROR: %s (err_num=%d)\n", hipGetErrorString(_cuda_tep_set_not_repeat_a), _cuda_tep_set_not_repeat_a); hipDeviceReset(); assert(0);} }

int main(){
    CUDA_CALL(hipSetDevice(0));
    //init
    const int length=1024;
    int a[length], b[length];
    for (int i=0;i<length;++i){
        a[i]=i;
        b[i]=i*i;
    }
    int *datas=NULL;
    hipMalloc((void **)&datas, length * sizeof(int));



    return 0;
}

