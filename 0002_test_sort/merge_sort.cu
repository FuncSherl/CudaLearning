#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cassert>
#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <algorithm>
#include <random>

using namespace std;

// 用宏变长参数来实现
#define CUDA_CALL(...) {hipError_t _cuda_tep_set_not_repeat_a=(__VA_ARGS__);if (_cuda_tep_set_not_repeat_a!=hipSuccess){printf("\nCUDA ERROR: %s (err_num=%d)\n", hipGetErrorString(_cuda_tep_set_not_repeat_a), _cuda_tep_set_not_repeat_a); hipDeviceReset(); assert(0);} }

__global__ void merge_sort(int *datas, int n){
    int tid=blockDim.x*threadIdx.y+threadIdx.x;
    extern __shared__ int shared[];
    if (tid<n) shared[tid] = datas[tid];
    __syncthreads();
    int cnt=1;
    for (int gap=2; gap<n*2; gap<<=1, cnt++){
        if (tid%gap==0){
            int left=tid+n*((cnt+1)%2);
            int mid=tid+gap/2+n*((cnt+1)%2);
            int right=mid;
            int end=tid+gap+((cnt+1)%2)*n;
            int full_end=(1+(cnt+1)%2)*n;
            int res_ind=n*(cnt%2)+tid;

            while((left<mid && left<full_end) || (right<end && right<full_end)){
                if (!(left<mid && left<full_end)){
                    shared[res_ind]=shared[right];
                    right++;
                }else if (!(right<end && right<full_end)){
                    shared[res_ind]=shared[left];
                    left++;
                }else{
                    if (shared[right]> shared[left]){
                        shared[res_ind]=shared[left];
                        left++;
                    }else{
                        shared[res_ind]=shared[right];
                        right++;
                    }
                }
                res_ind++;
            }           
        }
        __syncthreads();
    }

    datas[tid]=shared[tid+ ((cnt+1)%2)*n];
}

int main(){
    CUDA_CALL(hipSetDevice(0));
    //init
    const int length=1000;
    int a[length], b[length];
    for (int i=0;i<length;++i){
        a[i]=length-i;
        b[i]=i*i;
    }
    random_shuffle(begin(a), end(a));
    int *datas=NULL;
    CUDA_CALL(hipMalloc((void **)&datas, length * sizeof(int)));

    //hipError_t hipMemcpy ( void* dst, const void* src, size_t count,hipMemcpyKind kind )
    //hipMemcpyHostToHost   cudaMemcpyHossToDevice   hipMemcpyDeviceToHost   cudaMemcpuDeviceToDevice
    CUDA_CALL(hipMemcpy(datas,a,length*sizeof(int),hipMemcpyHostToDevice) );
    //注意这里传给kernel的shared mem大小是以字节度量的
    merge_sort<<<1, length, length*2*sizeof(int)>>>(datas, length);
    CUDA_CALL(hipGetLastError());

    CUDA_CALL( hipDeviceSynchronize());
    CUDA_CALL( hipMemcpy(a,datas,length*sizeof(int),hipMemcpyDeviceToHost));
    CUDA_CALL( hipFree(datas));

    for (int i=0;i<length;++i) cout<<a[i]<<" ";

    return 0;
}

