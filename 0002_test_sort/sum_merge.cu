#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include "hip/hip_runtime.h"
#include ""
#include <cassert>

using namespace std;

#define CUDA_CALL(x) {const hipError_t a=(x);if (a!=hipSuccess){printf("\nCUDA ERROR: %s (err_num=%d)\n", hipGetErrorString(a), a); hipDeviceReset(); assert(0);} }

//kernel function must return void
__global__ void do_sum_merge(int *datas, int n){
    int tid=blockDim.x*threadIdx.y+threadIdx.x;
    //int idx=blockIdx.x*blockDim.x+threadIdx.x;
    //int idy=blockIdx.y*blockDim.y+threadIdx.y;
    //int bid=gridDim.x*blockDim.x*idy+idx;
    while(n>1){
        if (tid< (1+n)/2 && n-1-tid!=tid){
            datas[tid]+=datas[n-1-tid];
            printf ("%d->%d->%d\n",n,tid,datas[tid]);             
        }
        n/=2;       
        __syncthreads();
    }
}

void cuda_call(hipError_t a){
    if (a!=hipSuccess){
        printf("\nCUDA ERROR: %s (err_num=%d)\n", hipGetErrorString(a), a); 
        hipDeviceReset(); 
        assert(0);
    }
}

int main(){
    CUDA_CALL(hipSetDevice(0));
    //init
    const int length=1024;
    int a[length], b[length];
    for (int i=0;i<length;++i){
        a[i]=1;
        b[i]=i;
    }
    int *datas=NULL;
    hipMalloc((void **)&datas, length * sizeof(int));
    //hipError_t hipMemcpy ( void* dst, const void* src, size_t count,hipMemcpyKind kind )
    //hipMemcpyHostToHost   cudaMemcpyHossToDevice   hipMemcpyDeviceToHost   cudaMemcpuDeviceToDevice
    cuda_call( hipMemcpy(datas,a,length*sizeof(int),hipMemcpyHostToDevice) );

    do_sum_merge<<<1,length>>>(datas,length);
    cuda_call(hipGetLastError());

    cuda_call( hipDeviceSynchronize());
    cuda_call( hipMemcpy(a,datas,length*sizeof(int),hipMemcpyDeviceToHost));
    hipFree(datas);

    for (int i=0;i<length;++i) cout<<a[i]<<" ";

    return 0;
}

