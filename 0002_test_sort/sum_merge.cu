#include "hip/hip_runtime.h"
#include <iostream>
#include "commontools.h"

using namespace std;

// kernel function must return void
__global__ void do_sum_merge(int *datas, int n) {
  int tid = blockDim.x * threadIdx.y + threadIdx.x;
  // int idx=blockIdx.x*blockDim.x+threadIdx.x;
  // int idy=blockIdx.y*blockDim.y+threadIdx.y;
  // int bid=gridDim.x*blockDim.x*idy+idx;
  while (n > 1) {
    if (tid < (1 + n) / 2 && n - 1 - tid != tid) {
      datas[tid] += datas[n - 1 - tid];
      printf("%d->%d->%d\n", n, tid, datas[tid]);
    }
    n /= 2;
    __syncthreads();
  }
}

void cuda_call(hipError_t a) {
  if (a != hipSuccess) {
    printf("\nCUDA ERROR: %s (err_num=%d)\n", hipGetErrorString(a), a);
    hipDeviceReset();
    assert(0);
  }
}

int main() {
  CUDA_CALL(hipSetDevice(0));
  // init
  const int length = 1024;
  int a[length];
  for (int i = 0; i < length; ++i) {
    a[i] = 1;
  }
  int *datas = NULL;
  CUDA_CALL(hipMalloc((void **)&datas, length * sizeof(int)));
  // hipError_t hipMemcpy ( void* dst, const void* src, size_t count,hipMemcpyKind kind )
  // hipMemcpyHostToHost   cudaMemcpyHossToDevice   hipMemcpyDeviceToHost   cudaMemcpuDeviceToDevice
  CUDA_CALL(hipMemcpy(datas, a, length * sizeof(int), hipMemcpyHostToDevice));

  do_sum_merge<<<1, length>>>(datas, length);
  CUDA_CALL(hipGetLastError());

  CUDA_CALL(hipDeviceSynchronize());
  CUDA_CALL(hipMemcpy(a, datas, length * sizeof(int), hipMemcpyDeviceToHost));
  CUDA_CALL(hipFree(datas));

  for (int i = 0; i < length; ++i) cout << a[i] << " ";

  return 0;
}
