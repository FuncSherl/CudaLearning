#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include "hip/hip_runtime.h"
#include ""

using namespace std;

#define CUDA_CALL(x) {const hipError_t a=(x);if (a!=hipSuccess){printf("\nCUDA ERROR: %s (err_num=%d)\n", hipGetErrorString(a), a); hipDeviceReset(); } }

//kernel function must return void
__global__ void do_sum_merge(int *datas, int n){
    int tid=blockDim.x*threadIdx.y+threadIdx.x;
    //int idx=blockIdx.x*blockDim.x+threadIdx.x;
    //int idy=blockIdx.y*blockDim.y+threadIdx.y;
    //int bid=gridDim.x*blockDim.x*idy+idx;
    if (tid< (1+n)/2){
        
    }
}

int main(){
    CUDA_CALL(hipSetDevice(0));
    //init
    const int length=1024;
    int a[length], b[length];
    for (int i=0;i<length;++i){
        a[i]=i;
        b[i]=i*i;
    }
    int *datas=NULL;
    CUDA_CALL(hipMalloc((void **)&datas, length * sizeof(int)));
    //hipError_t hipMemcpy ( void* dst, const void* src, size_t count,hipMemcpyKind kind )
    //hipMemcpyHostToHost   cudaMemcpyHossToDevice   hipMemcpyDeviceToHost   cudaMemcpuDeviceToDevice
    CUDA_CALL(hipMemcpy(datas, a, length * sizeof(int), cudaMemcpyHosToDevice));



    return 0;
}

