#include "hip/hip_runtime.h"
#define CUDA_CALL(x)                                                         \
    {                                                                        \
        const hipError_t a = (x);                                           \
        if (a != hipSuccess) {                                              \
            printf("\nCUDA ERROR: %s (err_num=%d)\n", hipGetErrorString(a), \
                   a);                                                       \
            hipDeviceReset();                                               \
            assert(0);                                                       \
        }                                                                    \
    }

#include <stdio.h>

#include "hip/hip_runtime.h"
#include ""

__global__ int do_sum_merge(int *datas){
    
}

int main(){
    CUDA_CALL(hipSetDevice(0));
    //init
    const int length=1024;
    int a[length], b[length];
    for (int i=0;i<length;++i){
        a[i]=i;
        b[i]=i*i;
    }
    int *datas=NULL;
    CUDA_CALL(hipMalloc((void **)&datas, length * sizeof(int)));
    //hipError_t hipMemcpy ( void* dst, const void* src, size_t count,hipMemcpyKind kind )
    //hipMemcpyHostToHost   cudaMemcpyHossToDevice   hipMemcpyDeviceToHost   cudaMemcpuDeviceToDevice
    CUDA_CALL(hipMemcpy(datas, a, length * sizeof(int), cudaMemcpyHossToDevice));


    return 0;
}

