#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <cstdlib>
#include <ctime>
#include <iostream>

#include "common/include/commontools.h"

#define THREAD_BLOCK_LIMIT 1024

template <typename T>
__global__ void cumsumBellochKernel(const T *d_in, T *d_out, long n) {
    extern __shared__ T temp[];
    long numPerThread = (n + blockDim.x - 1) / blockDim.x;
    long tid = threadIdx.x;
    long block_offset = blockIdx.x * n;

    // 1. copy the data to shared memory
    for (long i = 0; i < numPerThread; i++) {
        long index = tid * numPerThread + i;
        if (index < n) {
            temp[index] = d_in[block_offset + index];
        }
    }
    __syncthreads();

    // 2. local cumsum of each block
    for (long i = 1; i < numPerThread; i++) {
        long index = tid * numPerThread + i;
        if (index < n) {
            temp[index] += temp[index - 1];
        }
    }
    __syncthreads();

    // 3. Up-sweep phase (reduce)
    for (long stride = 1; stride < n; stride *= 2) {
        long stride_idx = stride * numPerThread;
        long index = (tid + 1) * stride_idx * 2 - 1;
        if (index < n) {
            temp[index] += temp[index - stride_idx];
        }
        __syncthreads();
    }

    // 4. Down-sweep phase (down)
    for (long stride = n / 4; stride > 0; stride /= 2) {
        long stride_idx = stride * numPerThread;
        long index = (tid + 1) * stride_idx * 2 - 1;
        if (index + stride_idx < n) {
            temp[index + stride_idx] += temp[index];
        }
        __syncthreads();
    }

    // 5. cumsum each local block
    for (long i = 0; i < numPerThread - 1; i++) {
        long index = (tid + 1) * numPerThread + i;
        if (index < n) {
            temp[index] += temp[(tid + 1) * numPerThread - 1];
        }
    }
    __syncthreads();

    // 6. copy the data back to global memory
    for (long i = 0; i < numPerThread; i++) {
        long index = tid * numPerThread + i;
        if (index < n) {
            d_out[block_offset + index] = temp[index];
        }
    }
}

template <typename T>
void cumsumBelloch(const T *h_in, T *h_out, int m, int n) {
    T *d_in, *d_out;
    size_t size = m * n * sizeof(T);

    CUDA_CALL(hipMalloc((void **)&d_in, size));
    CUDA_CALL(hipMalloc((void **)&d_out, size));

    CUDA_CALL(hipMemcpy(d_in, h_in, size, hipMemcpyHostToDevice));

    dim3 threadsPerBlock(std::min(THREAD_BLOCK_LIMIT, n));
    dim3 blocksPerGrid(m);
    clock_t start_kernel = clock();
    cumsumBellochKernel<<<blocksPerGrid, threadsPerBlock, n * sizeof(T)>>>(
        d_in, d_out, n);
    CUDA_LAST_ERROR();
    CUDA_CALL(hipDeviceSynchronize());
    clock_t end_kernel = clock();
    double elapsed_time_kernel =
        double(end_kernel - start_kernel) * 1000.0 / CLOCKS_PER_SEC;
    std::cout << "Kernel cumsumBelloch execution time: " << elapsed_time_kernel
              << " milliseconds" << std::endl;

    CUDA_CALL(hipMemcpy(h_out, d_out, size, hipMemcpyDeviceToHost));

    CUDA_CALL(hipFree(d_in));
    CUDA_CALL(hipFree(d_out));
}

template <typename T>
__global__ void cumsumNaiveKernel(const T *d_in, T *d_out, int n) {
    extern __shared__ T temp[];
    long bufferOffset = n * sizeof(T);
    long numPerThread = (n + blockDim.x - 1) / blockDim.x;
    long tid = threadIdx.x;
    long block_offset = blockIdx.x * n;

    // 1. copy the data to shared memory
    for (long i = 0; i < numPerThread; i++) {
        long index = tid * numPerThread + i;
        if (index < n) {
            temp[index] = d_in[block_offset + index];
        }
    }
    __syncthreads();

    // 2. naive tree sum
    int cnt = 0;
    for (long stride = 1; stride < n; stride *= 2, cnt++) {
        int readIdx = cnt % 2;
        int writeIdx = 1 - readIdx;
        T *tempR = temp + readIdx * bufferOffset;
        T *tempW = temp + writeIdx * bufferOffset;

        for (long j = 0; j < numPerThread; j++) {
            long index = tid * numPerThread + j;

            if (index >= stride && index < n) {
                tempW[index] = tempR[index] + tempR[index - stride];
            } else {
                tempW[index] = tempR[index];
            }
        }
        __syncthreads();
    }

    // 3. copy the data back to global memory
    for (long i = 0; i < numPerThread; i++) {
        long index = tid * numPerThread + i;
        if (index < n) {
            d_out[block_offset + index] =
                temp[index + (cnt % 2) * bufferOffset];
        }
    }
}

template <typename T>
void cumsumNaive(const T *h_in, T *h_out, int m, int n) {
    T *d_in, *d_out;
    size_t size = m * n * sizeof(T);

    CUDA_CALL(hipMalloc((void **)&d_in, size));
    CUDA_CALL(hipMalloc((void **)&d_out, size));

    CUDA_CALL(hipMemcpy(d_in, h_in, size, hipMemcpyHostToDevice));

    dim3 threadsPerBlock(std::min(THREAD_BLOCK_LIMIT, n));
    dim3 blocksPerGrid(m);
    clock_t start_kernel = clock();
    cumsumNaiveKernel<<<blocksPerGrid, threadsPerBlock, n * sizeof(T) * 2>>>(
        d_in, d_out, n);
    CUDA_LAST_ERROR();
    CUDA_CALL(hipDeviceSynchronize());
    clock_t end_kernel = clock();
    double elapsed_time_kernel =
        double(end_kernel - start_kernel) * 1000.0 / CLOCKS_PER_SEC;
    std::cout << "Kernel cumsumNaive execution time: " << elapsed_time_kernel
              << " milliseconds" << std::endl;

    CUDA_CALL(hipMemcpy(h_out, d_out, size, hipMemcpyDeviceToHost));

    CUDA_CALL(hipFree(d_in));
    CUDA_CALL(hipFree(d_out));
}

void cumsumSingle(const int *h_in, int *h_out, int m, int n) {
    clock_t start = clock();
    for (int i = 0; i < m; i++) {
        h_out[i * n] = h_in[i * n];
        for (int j = 1; j < n; j++) {
            h_out[i * n + j] = h_out[i * n + j - 1] + h_in[i * n + j];
        }
    }
    clock_t end = clock();
    double elapsed_time = double(end - start) * 1000.0 / CLOCKS_PER_SEC;
    std::cout << "Single cumsum execution time: " << elapsed_time
              << " milliseconds" << std::endl;
}

bool checkDiff(const int *a, const int *b, long size) {
    for (long i = 0; i < size; i++) {
        if (a[i] != b[i]) {
            std::cout << "Difference found at index " << i << ": " << a[i]
                      << " != " << b[i] << std::endl;
            return false;
        }
    }
    std::cout << "No differences found." << std::endl;
    return true;
}

int main(int argc, char *argv[]) {
    if (argc != 3) {
        std::cerr << "Usage: " << argv[0] << " <m> <n>" << std::endl;
        return 1;
    }

    const long m = std::stol(argv[1]);
    const long n = std::stol(argv[2]);

    int *h_in = new int[m * n];
    int *h_out_cpu = new int[m * n];
    int *h_out = new int[m * n];

    srand(time(0));

    std::cout << "Initializing input array..." << std::endl;
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            h_in[i * n + j] = rand() % 10;  // Random numbers between 0 and 9
        }
    }

    std::cout << "Running single-threaded cumulative sum..." << std::endl;
    cumsumSingle(h_in, h_out_cpu, m, n);

    std::cout << "Running naive parallel cumulative sum..." << std::endl;
    cumsumNaive(h_in, h_out, m, n);
    std::cout << "Checking results of naive parallel cumulative sum..."
              << std::endl;
    checkDiff(h_out, h_out_cpu, m * n);

    std::fill(h_out, h_out + m * n, 0);

    std::cout << "Running Belloch parallel cumulative sum..." << std::endl;
    cumsumBelloch(h_in, h_out, m, n);
    std::cout << "Checking results of Belloch parallel cumulative sum..."
              << std::endl;
    checkDiff(h_out, h_out_cpu, m * n);

    delete[] h_in;
    delete[] h_out_cpu;
    delete[] h_out;

    return 0;
}
