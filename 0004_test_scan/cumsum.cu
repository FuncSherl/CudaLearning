#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <cmath>
#include <cstdlib>
#include <ctime>
#include <iostream>

#include "common/include/commontools.h"

#define THREAD_BLOCK_LIMIT 1024

// Cumsum with belloch parallelization
template <typename T>
__global__ void cumsumBellochKernel(const T *d_in, T *d_out, long n) {
    extern __shared__ T temp[];
    assert((blockDim.x & (blockDim.x - 1)) == 0 &&
           "blockDim.x must be a power of 2");

    long remain = n % blockDim.x;
    long numPerThread = n / blockDim.x;
    long tid = threadIdx.x;
    // Calculate the starting index for each thread and adjust the number of
    // elements per thread
    long indexStart = tid * numPerThread + min(tid, remain);
    long numThisThread = numPerThread + ((tid < remain) ? 1 : 0);

    long block_offset = blockIdx.x * n;

    // 1. copy the data to shared memory
    for (long i = 0; i < numThisThread; i++) {
        long index = indexStart + i;
        if (index < n) {
            temp[index] = d_in[block_offset + index];
        }
    }
    __syncthreads();

    // 2. local cumsum of each block
    for (long i = 1; i < numThisThread; i++) {
        long index = indexStart + i;
        if (index < n) {
            temp[index] += temp[index - 1];
        }
    }
    __syncthreads();

    // 3. Up-sweep phase (reduce)
    for (long stride = 1; stride < n; stride *= 2) {
        long index = (tid + 1) * stride * 2 - 1;
        long indexSt = index * numPerThread + min(index, remain) +
                       numPerThread + (index < remain) - 1;
        long indexEd = (index - stride) * numPerThread +
                       min((index - stride), remain) + numPerThread +
                       ((index - stride) < remain) - 1;

        if (indexSt < n) {
            temp[indexSt] += temp[indexEd];
        }
        __syncthreads();
    }

    // 4. Down-sweep phase (down)
    for (long stride = n / 4; stride > 0; stride /= 2) {
        long index = (tid + 1) * stride * 2 - 1;
        long indexSt = index * numPerThread + min(index, remain) +
                       numPerThread + (index < remain) - 1;
        long indexEd = (index + stride) * numPerThread +
                       min((index + stride), remain) + numPerThread +
                       ((index + stride) < remain) - 1;
        if (indexEd < n) {
            temp[indexEd] += temp[indexSt];
        }
        __syncthreads();
    }

    // 5. cumsum each local block
    long tidAddone = numPerThread + ((tid + 1 < remain) ? 1 : 0);
    for (long i = 0; i < tidAddone - 1; i++) {
        long indexSt = (tid + 1) * numPerThread + min(tid + 1, remain);
        long index = indexSt + i;
        if (index < n) {
            temp[index] += temp[indexSt - 1];
        }
    }
    __syncthreads();

    // 6. copy the data back to global memory
    for (long i = 0; i < numThisThread; i++) {
        long index = indexStart + i;
        if (index < n) {
            d_out[block_offset + index] = temp[index];
        }
    }
}

template <typename T>
void cumsumBelloch(const T *h_in, T *h_out, int m, int n) {
    T *d_in, *d_out;
    size_t size = m * n * sizeof(T);

    CUDA_CALL(hipMalloc((void **)&d_in, size));
    CUDA_CALL(hipMalloc((void **)&d_out, size));

    CUDA_CALL(hipMemcpy(d_in, h_in, size, hipMemcpyHostToDevice));
    // Calculate the next power of 2 greater than or equal to n
    int n_pow2 = 1 << static_cast<int>(log2(n));
    dim3 threadsPerBlock(std::min(THREAD_BLOCK_LIMIT, n_pow2));
    dim3 blocksPerGrid(m);
    std::cout << "Launching kernel with " << blocksPerGrid.x << " blocks and "
              << threadsPerBlock.x << " threads per block." << std::endl;
    clock_t start_kernel = clock();
    cumsumBellochKernel<<<blocksPerGrid, threadsPerBlock, n * sizeof(T)>>>(
        d_in, d_out, n);
    CUDA_LAST_ERROR();
    CUDA_CALL(hipDeviceSynchronize());
    clock_t end_kernel = clock();
    double elapsed_time_kernel =
        double(end_kernel - start_kernel) * 1000.0 / CLOCKS_PER_SEC;
    std::cout << "Kernel GPU cumsumBelloch execution time for m = " << m
              << ", n = " << n << ": " << elapsed_time_kernel << " milliseconds"
              << std::endl;

    CUDA_CALL(hipMemcpy(h_out, d_out, size, hipMemcpyDeviceToHost));

    CUDA_CALL(hipFree(d_in));
    CUDA_CALL(hipFree(d_out));
}

// Cumsum with navie parallelization
template <typename T>
__global__ void cumsumNaiveKernel(const T *d_in, T *d_out, int n) {
    extern __shared__ T temp[];
    long numPerThread = (n + blockDim.x - 1) / blockDim.x;
    long tid = threadIdx.x;
    long block_offset = blockIdx.x * n;

    // 1. copy the data to shared memory
    for (long i = 0; i < numPerThread; i++) {
        long index = tid * numPerThread + i;
        if (index < n) {
            temp[index] = d_in[block_offset + index];
        }
    }
    __syncthreads();

    // 2. naive tree sum
    int cnt = 0;
    for (long stride = 1; stride < n; stride *= 2, cnt++) {
        int readIdx = cnt % 2;
        int writeIdx = 1 - readIdx;
        T *tempR = temp + readIdx * n;
        T *tempW = temp + writeIdx * n;

        for (long j = 0; j < numPerThread; j++) {
            long index = tid * numPerThread + j;
            if (index < n) {
                if (index >= stride) {
                    tempW[index] = tempR[index] + tempR[index - stride];
                } else {
                    tempW[index] = tempR[index];
                }
            }
        }
        __syncthreads();
    }

    // 3. copy the data back to global memory
    for (long i = 0; i < numPerThread; i++) {
        long index = tid * numPerThread + i;
        if (index < n) {
            d_out[block_offset + index] = temp[index + (cnt % 2) * n];
        }
    }
}

template <typename T>
void cumsumNaive(const T *h_in, T *h_out, int m, int n) {
    T *d_in, *d_out;
    size_t size = m * n * sizeof(T);

    CUDA_CALL(hipMalloc((void **)&d_in, size));
    CUDA_CALL(hipMalloc((void **)&d_out, size));

    CUDA_CALL(hipMemcpy(d_in, h_in, size, hipMemcpyHostToDevice));

    dim3 threadsPerBlock(std::min(THREAD_BLOCK_LIMIT, n));
    dim3 blocksPerGrid(m);
    clock_t start_kernel = clock();
    cumsumNaiveKernel<<<blocksPerGrid, threadsPerBlock, n * sizeof(T) * 2>>>(
        d_in, d_out, n);
    CUDA_LAST_ERROR();
    CUDA_CALL(hipDeviceSynchronize());
    clock_t end_kernel = clock();
    double elapsed_time_kernel =
        double(end_kernel - start_kernel) * 1000.0 / CLOCKS_PER_SEC;
    std::cout << "Kernel GPU cumsumNaive execution time for m = " << m
              << ", n = " << n << ": " << elapsed_time_kernel << " milliseconds"
              << std::endl;

    CUDA_CALL(hipMemcpy(h_out, d_out, size, hipMemcpyDeviceToHost));

    CUDA_CALL(hipFree(d_in));
    CUDA_CALL(hipFree(d_out));
}

// Cumsum with single thread
template <typename T>
__global__ void cumsumSingleKernel(const T *d_in, T *d_out, int n) {
    extern __shared__ T temp[];
    long numPerThread = (n + blockDim.x - 1) / blockDim.x;
    long tid = threadIdx.x;
    long block_offset = blockIdx.x * n;
    // 1. copy the data to shared memory
    for (long i = 0; i < numPerThread; i++) {
        long index = tid * numPerThread + i;
        if (index < n) {
            temp[index] = d_in[block_offset + index];
        }
    }
    __syncthreads();

    // 2. cumsum with only one thread
    if (tid == 0) {
        for (long i = 1; i < n; i++) {
            temp[i] += temp[i - 1];
        }
    }
    __syncthreads();

    // 3. copy the data back to global memory
    for (long i = 0; i < numPerThread; i++) {
        long index = tid * numPerThread + i;
        if (index < n) {
            d_out[block_offset + index] = temp[index];
        }
    }
}

template <typename T>
void cumsumSingle(const T *h_in, T *h_out, int m, int n) {
    T *d_in, *d_out;
    size_t size = m * n * sizeof(T);

    CUDA_CALL(hipMalloc((void **)&d_in, size));
    CUDA_CALL(hipMalloc((void **)&d_out, size));

    CUDA_CALL(hipMemcpy(d_in, h_in, size, hipMemcpyHostToDevice));

    dim3 threadsPerBlock(std::min(THREAD_BLOCK_LIMIT, n));
    dim3 blocksPerGrid(m);
    clock_t start_kernel = clock();
    cumsumSingleKernel<<<blocksPerGrid, threadsPerBlock, n * sizeof(T)>>>(
        d_in, d_out, n);
    CUDA_LAST_ERROR();
    CUDA_CALL(hipDeviceSynchronize());
    clock_t end_kernel = clock();
    double elapsed_time_kernel =
        double(end_kernel - start_kernel) * 1000.0 / CLOCKS_PER_SEC;
    std::cout << "Kernel GPU cumsumSingle execution time for m = " << m
              << ", n = " << n << ": " << elapsed_time_kernel << " milliseconds"
              << std::endl;

    CUDA_CALL(hipMemcpy(h_out, d_out, size, hipMemcpyDeviceToHost));

    CUDA_CALL(hipFree(d_in));
    CUDA_CALL(hipFree(d_out));
}

// Cumsum with CPU single thread as golden
template <typename T>
void cumsumGolden(const T *h_in, T *h_out, int m, int n) {
    clock_t start = clock();
    for (int i = 0; i < m; i++) {
        h_out[i * n] = h_in[i * n];
        for (int j = 1; j < n; j++) {
            h_out[i * n + j] = h_out[i * n + j - 1] + h_in[i * n + j];
        }
    }
    clock_t end = clock();
    double elapsed_time = double(end - start) * 1000.0 / CLOCKS_PER_SEC;
    std::cout << "Single CPU cumsum execution time for m = " << m
              << ", n = " << n << ": " << elapsed_time << " milliseconds"
              << std::endl;
}

template <typename T>
bool checkDiff(const T *a, const T *b, long size) {
    for (long i = 0; i < size; i++) {
        if (a[i] != b[i]) {
            if constexpr (std::is_same<T, char>::value) {
                std::cout << "Difference found at index " << i << ": "
                          << static_cast<int>(a[i])
                          << " != " << static_cast<int>(b[i]) << std::endl;
            } else {
                std::cout << "Difference found at index " << i << ": " << a[i]
                          << " != " << b[i] << std::endl;
            }
            return false;
        }
    }
    std::cout << "No differences found." << std::endl;
    return true;
}

int main(int argc, char *argv[]) {
    if (argc != 3) {
        std::cerr << "Usage: " << argv[0] << " <m> <n>" << std::endl;
        return 1;
    }

    const long m = std::stol(argv[1]);
    const long n = std::stol(argv[2]);

    using dtype = char;
    dtype *h_in = new dtype[m * n];
    dtype *h_out_cpu = new dtype[m * n];
    dtype *h_out = new dtype[m * n];

    srand(time(0));

    std::cout << "Initializing input array..." << std::endl;
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            h_in[i * n + j] = rand() % 10;  // Random numbers between 0 and 9
        }
    }
    // Call cpu golden
    std::cout << "\nRunning CPU single-threaded cumulative sum..." << std::endl;
    cumsumGolden(h_in, h_out_cpu, m, n);

    // Call single-threaded GPU cumsum
    std::cout << "\nRunning single-threaded GPU cumulative sum..." << std::endl;
    cumsumSingle(h_in, h_out, m, n);
    std::cout << "Checking results of single-threaded GPU cumulative sum..."
              << std::endl;
    checkDiff(h_out, h_out_cpu, m * n);

    // refill h_out
    std::fill(h_out, h_out + m * n, 0);

    // Call naive
    std::cout << "\nRunning naive parallel cumulative sum..." << std::endl;
    cumsumNaive(h_in, h_out, m, n);
    std::cout << "Checking results of naive parallel cumulative sum..."
              << std::endl;
    checkDiff(h_out, h_out_cpu, m * n);

    // refill h_out
    std::fill(h_out, h_out + m * n, 0);

    // Call belloch
    std::cout << "\nRunning Belloch parallel cumulative sum..." << std::endl;
    cumsumBelloch(h_in, h_out, m, n);
    std::cout << "Checking results of Belloch parallel cumulative sum..."
              << std::endl;
    checkDiff(h_out, h_out_cpu, m * n);

    delete[] h_in;
    delete[] h_out_cpu;
    delete[] h_out;

    return 0;
}
